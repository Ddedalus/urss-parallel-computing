#include "hip/hip_runtime.h"


// MutIn.cu:  finds mean number of mutual inlinks, 
// among all pairs of Web sites in our set; in checking 
// (i,j) pairs, thread k will handle all i such that 
// i mod totth = k, where totth is the number of threads

// usage:
//
//    mutin numvertices numblocks

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// block size is hard coded as 192
#define BLOCKSIZE 192

// kernel:  processes all pairs assigned to a given thread
__global__ void procpairs(int *m, int *tot, int n)
{
   // total number of threads = number of blocks * block size
   int totth = gridDim.x * BLOCKSIZE,  
       // my thread number
       me = blockIdx.x * blockDim.x + threadIdx.x;  
   int i,j,k,sum = 0; 
   for (i = me; i < n; i += totth) {  // various columns i
      for (j = i+1; j < n; j++) {  // all columns j > i
         for (k = 0; k < n; k++)
            sum += m[n*k+i] * m[n*k+j];
      }
   }
   atomicAdd(tot,sum);
}

int main(int argc, char **argv)
{  int n = atoi(argv[1]),  // number of vertices
       nblk = atoi(argv[2]);  // number of blocks
    // the usual initializations
    int *hm, // host matrix
        *dm, // device matrix
        htot, // host grand total
        *dtot; // device grand total
    int msize = n * n * sizeof(int);  
    hm = (int *) malloc(msize);  
    // as a test, fill matrix with random 1s and 0s
    int i,j;
    for (i = 0; i < n; i++) {
       hm[n*i+i] = 0;
       for (j = 0; j < n; j++) {
          if (j != i) hm[i*n+j] = rand() % 2;
       }
    }
    // more of the usual initializations
    hipMalloc((void **)&dm,msize);
    // copy host matrix to device matrix
    hipMemcpy(dm,hm,msize,hipMemcpyHostToDevice);
    htot = 0;
    // set up device total and initialize it
    hipMalloc((void **)&dtot,sizeof(int));
    hipMemcpy(dtot,&htot,sizeof(int),hipMemcpyHostToDevice);
    // OK, ready to launch kernel, so configure grid
    dim3 dimGrid(nblk,1);
    dim3 dimBlock(BLOCKSIZE,1,1);
    // launch the kernel
    procpairs<<<dimGrid,dimBlock>>>(dm,dtot,n);
    // wait for kernel to finish
    hipDeviceSynchronize();
    // copy total from device to host
    hipMemcpy(&htot,dtot,sizeof(int),hipMemcpyDeviceToHost);
    // check results
    if (n <= 15) {
       for (i = 0; i < n; i++) {
          for (j = 0; j < n; j++) 
             printf("%d ",hm[n*i+j]);
          printf("\n");
       }
    }
    printf("mean = %f\n",htot/(float)((n*(n-1))/2));
    // clean up
    free(hm);
    hipFree(dm);
    hipFree(dtot);
}
